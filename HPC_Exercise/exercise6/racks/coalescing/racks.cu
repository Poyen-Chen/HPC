#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "realtime.h"
// 4096*4096 // 8192*8192
#define N 67108864-1024 // 16777216 //67108864 
#define THREADSPERBLOCK 1024

// struct (AoS)
struct rack_t {
	float widthA;
	float widthB;
	float doubledWidth;
};

// struct of arrays (SoA)
struct rackSoA_t {
	float *widthA;
	float *widthB;
	float *doubledWidth;
};

static void initGPU(int argc, char** argv);
static void initRacks(rack_t *racks, int n);
static void initRacksSoA(rackSoA_t *racks, int n);


#ifdef WIN32
__inline void checkErr(hipError_t err, const char* file, const int line);
#else
inline void checkErr(hipError_t err, const char* file, const int line);
#endif

// GPU kernel: Array of Structures (AoS)
__global__ void doubleTheWidth(rack_t *racks, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < n) {
		racks[tid].doubledWidth = 2 * (racks[tid].widthA + racks[tid].widthB);
	}
}

// GPU kernel: Structure of Arrays (SoA)
__global__ void doubleTheWidthSoA(rackSoA_t racks, int n)
{
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid < n) {
                racks.doubledWidth[tid] = 2 * (racks.widthA[tid] + racks.widthB[tid]);
        }
}

int main(int argc, char** argv)
{
    initGPU(argc, argv);


    const int n = N;
    hipError_t err;
    double runtimeAll, runtimeKernel;
   
    //#######################
    //# ARRAY OF STRUCTURES #
    //####################### 

    printf("Array of Structures\n");

    rack_t *h_racks = 0;
    rack_t *d_racks = 0;

    // allocate memory
    h_racks = (rack_t*) malloc(n*sizeof(rack_t));
    if (h_racks == 0) { printf("Not enough memory\n.");}
    err = hipMalloc((void**)&d_racks,n*sizeof(rack_t));
    checkErr(err, __FILE__, __LINE__);

    // init racks struct
    initRacks(h_racks,n);
    printf("First rack: w1=%f, w2=%f\n",h_racks[0].widthA, h_racks[0].widthB);

    runtimeAll = GetRealTime();

    // copy to GPU
    err = hipMemcpy(d_racks,h_racks,n*sizeof(rack_t),hipMemcpyHostToDevice);
    checkErr(err, __FILE__, __LINE__);

    dim3 threads_per_block(THREADSPERBLOCK);
    dim3 blocks_per_grid;

    // Compute the number of blocks_per_grid
    blocks_per_grid = dim3((n+(THREADSPERBLOCK-1))/THREADSPERBLOCK);
    printf("blocks: %d\n",blocks_per_grid.x);

    runtimeKernel = GetRealTime();
    doubleTheWidth<<<blocks_per_grid,threads_per_block>>>(d_racks,n);
    hipDeviceSynchronize();
    runtimeKernel = GetRealTime() - runtimeKernel;

    err = hipMemcpy(h_racks,d_racks,n*sizeof(rack_t),hipMemcpyDeviceToHost);
    checkErr(err, __FILE__, __LINE__);
    
    runtimeAll = GetRealTime() - runtimeAll;

    printf("First rack: doubled width=%f\n",h_racks[0].doubledWidth);
    printf("Time Elapsed (including data transfer): %f s\n", runtimeAll);
    printf("Time Elapsed (kernel): %f s\n", runtimeKernel);

    //#######################
    //# STRUCTURE OF ARRAYS #
    //#######################

    printf("\nStructure of Arrays\n");

    rackSoA_t h_racksSoA;
    rackSoA_t d_racksSoA;

    // allocate memory
    h_racksSoA.widthA = (float*) malloc(n*sizeof(float));
    h_racksSoA.widthB = (float*) malloc(n*sizeof(float));
    h_racksSoA.doubledWidth = (float*) malloc(n*sizeof(float));
    if (h_racksSoA.widthA == 0 || h_racksSoA.widthB == 0 || h_racksSoA.doubledWidth == 0) { printf("Not enough memory\n.");}
    err = hipMalloc((void**)&d_racksSoA.widthA,n*sizeof(float));
    checkErr(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_racksSoA.widthB,n*sizeof(float));
    checkErr(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_racksSoA.doubledWidth,n*sizeof(float));
    checkErr(err, __FILE__, __LINE__);

    // init racks struct
    initRacksSoA(&h_racksSoA,n);
    printf("First rack: w1=%f, w2=%f\n",h_racksSoA.widthA[0], h_racksSoA.widthB[0]);

    runtimeAll = GetRealTime();

    // copy to GPU
    err = hipMemcpy(d_racksSoA.widthA,h_racksSoA.widthA,n*sizeof(float),hipMemcpyHostToDevice);
    checkErr(err, __FILE__, __LINE__);
    err = hipMemcpy(d_racksSoA.widthB,h_racksSoA.widthB,n*sizeof(float),hipMemcpyHostToDevice);
    checkErr(err, __FILE__, __LINE__);
    // Transfer of doubledWidth not needed since values are created on GPU
    //err = hipMemcpy(d_racksSoA.doubledWidth,h_racksSoA.doubledWidth,n*sizeof(float),hipMemcpyHostToDevice);
    //checkErr(err, __FILE__, __LINE__);

    runtimeKernel = GetRealTime();
    doubleTheWidthSoA<<<blocks_per_grid,threads_per_block>>>(d_racksSoA,n);
    hipDeviceSynchronize();
    runtimeKernel = GetRealTime() - runtimeKernel;

    // Transfer of widthA and widthB not needed since not modified
    //err = hipMemcpy(h_racksSoA.widthA,d_racksSoA.widthA,n*sizeof(float),hipMemcpyDeviceToHost);
    //checkErr(err, __FILE__, __LINE__);
    //err = hipMemcpy(h_racksSoA.widthB,d_racksSoA.widthB,n*sizeof(float),hipMemcpyDeviceToHost);
    //checkErr(err, __FILE__, __LINE__);
    err = hipMemcpy(h_racksSoA.doubledWidth,d_racksSoA.doubledWidth,n*sizeof(float),hipMemcpyDeviceToHost);
    checkErr(err, __FILE__, __LINE__);

    runtimeAll = GetRealTime() - runtimeAll;

    printf("First rack: doubled width=%f\n",h_racksSoA.doubledWidth[0]);
    printf("Time Elapsed (including data transfer): %f s\n", runtimeAll);
    printf("Time Elapsed (kernel): %f s\n", runtimeKernel);

    free(h_racks);
    free(h_racksSoA.widthA);
    free(h_racksSoA.widthB);
    free(h_racksSoA.doubledWidth);
    hipFree(d_racks);
    hipFree(d_racksSoA.widthA);
    hipFree(d_racksSoA.widthB);
    hipFree(d_racksSoA.doubledWidth);
    return 0;
}

static void initRacks(rack_t *racks, int n) {
    for(int i=0; i<n; i++) {
	racks[i].widthA = i+2.5;
	racks[i].widthB = i+1.5;
    }
}

static void initRacksSoA(rackSoA_t *racks, int n) {
    for(int i=0; i<n; i++) {
        (*racks).widthA[i] = i+2.5;
        (*racks).widthB[i] = i+1.5;
    }
}

static void initGPU(int argc, char** argv) {
        // gets the device id (if specified) to run on
        int devId = -1;
        int devCount = 0;
        if (argc > 1) {
                devId = atoi(argv[1]);
                hipGetDeviceCount(&devCount);
                if (devId < 0 || devId >= devCount) {
                        printf("The specified device ID is not supported.\n");
                        exit(1);
                }
        }
        if (devId != -1) {
                hipSetDevice(devId);
        }
        // creates a context on the GPU just to
        // exclude initialization time from computations
        hipFree(0);

        // print device id
        hipGetDevice(&devId);
        printf("Running on GPU with ID %d.\n\n", devId);

}


// Checks whether a CUDA error occured
// If so, the error message is printed and the program exits
inline void checkErr(hipError_t err, const char* file, const int line)
{
        if(hipSuccess != err)
        {
                fprintf(stderr, "%s: Cuda error in line %d: %s.\n", file, line, hipGetErrorString(err) );
                exit(-1);
        }
}


