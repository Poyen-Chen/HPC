#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include "realtime.h"
#include <stdio.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#define ROWS 4096
#define COLS 4096

double U[ROWS][COLS];
double Unew[ROWS][COLS];

__global__
void laplace2DKernel(double *error, double* Unew, double* U, int n, int m) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if( i > 0 && i < n-1 && j > 0 && j < m-1) {
    	//Unew[i][j] = 0.25 * ( U[i][j+1] + U[i][j-1] + U[i-1][j] + U[i+1][j]);
        //error[j*n+1] = fabs(Unew[i][j] - U[i][j]);
    Unew[j*n+i] = 0.25 * ( U[j * n + i+1] + U[j* n + i -1]	
			  + U[(j-1) * n + i] + U[(j+1)* n + i]);

    error[j*n+i] = fabs(Unew[j*n+i] - U[j*n+i]);
  }

}

__global__
void swapKernel(double* Unew, double* U, int n, int m) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  
  if( i > 0 && i < n-1 && j > 0 && j < m-1) {  
    U[j*n+i] = Unew[j*n+i];
  }
}

int main(int argc, char** argv)
{
    const int n = ROWS;
    const int m = COLS;
    const int iter_max = 20;
    
    const double tol = 1.0e-6;
    double err       = 1.0;
    
    // Initialize arrays
    memset(U, 0, n * m * sizeof(double));
    memset(Unew, 0, n * m * sizeof(double));        
    for (int i = 0; i < n; i++)
    {
        U[0][i]    = -1.0;
        Unew[0][i] = -1.0;
    }

    double* dU;
    double* dUnew;
    double* dError;

    hipMalloc(&dU, n*m*sizeof(double));
    hipMalloc(&dUnew, n*m*sizeof(double));
    hipMalloc(&dError, n*m*sizeof(double));
    
    printf("Solving Laplace Equation by Jacobi Method\n");
    printf("Matrix dim: %d x %d\n\n", n, m);
    printf("Iteration: Error\n");    

    int iter = 0;
    
    dim3 block(64,4);
    dim3 grid(n/block.x, m/block.y);


    double runtime = GetRealTime();
    hipMemcpy(dU,U,n*m*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dUnew,Unew,n*m*sizeof(double), hipMemcpyHostToDevice);
    
    while ( err > tol && iter < iter_max )
    {
      hipMemsetAsync(dError, 0,  n*m*sizeof(double));

      laplace2DKernel<<<grid,block>>>(dError, dUnew, dU, n, m);      

      //hipMemcpyAsync(dU,dUnew, n*m*sizeof(double), hipMemcpyDeviceToDevice); // Pointer swap: not comparable to other versions
      // instead:
      swapKernel<<<grid,block>>>(dUnew, dU, n, m);
      
      thrust::device_ptr<double> thrust_error = thrust::device_pointer_cast(dError);
      err = thrust::reduce(thrust_error, thrust_error + n*m, 0.0, thrust::maximum<double>());
      
      iter++;
      
      printf("%9d: %f\n", iter, err);
    }
    
    runtime = GetRealTime() - runtime;
    
    printf("Time Elapsed: %f s\n", runtime);
 
    hipFree(dU);
    hipFree(dUnew);
    hipFree(dError);

    return 0;
}

